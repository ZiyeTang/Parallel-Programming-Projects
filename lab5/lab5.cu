#include "hip/hip_runtime.h"
// MP5 Reduction
// Input: A num list of length n
// Output: Sum of the list = list[0] + list[1] + ... + list[n-1];

#include <wb.h>

#define BLOCK_SIZE 512 //@@ This value is not fixed and you can adjust it according to the situation

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)
  
__global__ void total(float *input, float *output, int len) {
  //@@ Load a segment of the input vector into shared memory
  __shared__ float partialSum[2 * BLOCK_SIZE];

  unsigned int start = 2*blockIdx.x*blockDim.x;
  unsigned int t = threadIdx.x;
  if (start+t < len) {
    partialSum[t] = input[start + t];
  } else {
    partialSum[t] = 0;
  }
  
  if (start+blockDim.x+t<len) {
    partialSum[blockDim.x+t] = input[start+ blockDim.x+t];
  } else {
    partialSum[blockDim.x+t] = 0;
  }
  

  //@@ Traverse the reduction tree
  for (unsigned int stride = blockDim.x; stride >= 1; stride /= 2) {
    __syncthreads();
    if (t < stride)
      partialSum[t] += partialSum[t+stride];
  }

  //@@ Write the computed sum of the block to the output vector at the correct index
  output[blockIdx.x] = partialSum[0];
}

int main(int argc, char **argv) {
  int ii;
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  //@@ Initialize device input and output pointers
  float *deviceInput, *deviceOutput;

  int numInputElements;  // number of elements in the input list
  int numOutputElements; // number of elements in the output list

  args = wbArg_read(argc, argv);

  //Import data and create memory on host
  hostInput =
      (float *)wbImport(wbArg_getInputFile(args, 0), &numInputElements);

  numOutputElements = numInputElements / (BLOCK_SIZE << 1);
  if (numInputElements % (BLOCK_SIZE << 1)) {
    numOutputElements++;
  }
  hostOutput = (float *)malloc(numOutputElements * sizeof(float));

  // The number of input elements in the input is numInputElements
  // The number of output elements in the input is numOutputElements

  //@@ Allocate GPU memory
  int inputSize = sizeof(float) * numInputElements;
  int outputSize = sizeof(float) * numOutputElements;
  hipMalloc((void **) &deviceInput, inputSize);
  hipMalloc((void **) &deviceOutput, outputSize);

  //@@ Copy input memory to the GPU
  hipMemcpy(deviceInput, hostInput, inputSize, hipMemcpyHostToDevice);

  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(numOutputElements, 1, 1);
  dim3 DimBlock(BLOCK_SIZE, 1, 1);

  //@@ Launch the GPU Kernel and perform CUDA computation
  total<<<DimGrid,DimBlock>>>(deviceInput, deviceOutput, numInputElements);
  
  hipDeviceSynchronize();  
  //@@ Copy the GPU output memory back to the CPU
  hipMemcpy(hostOutput, deviceOutput, outputSize, hipMemcpyDeviceToHost);
  
  /********************************************************************
   * Reduce output vector on the host
   * NOTE: One could also perform the reduction of the output vector
   * recursively and support any size input. 
   * For simplicity, we do not require that for this lab.
   ********************************************************************/
  for (ii = 1; ii < numOutputElements; ii++) {
    hostOutput[0] += hostOutput[ii];
  }

  //@@ Free the GPU memory
  hipFree(deviceInput);
  hipFree(deviceOutput);


  wbSolution(args, hostOutput, 1);

  free(hostInput);
  free(hostOutput);

  return 0;
}

